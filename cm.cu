#include "hip/hip_runtime.h"
/*
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <thrust/set_operations.h>
#include <thrust/gather.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/adjacent_difference.h>
#include <thrust/transform.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <sstream>
#include <stdio.h>
#include <fstream>
#include <iomanip>
#include <queue>
#include <set>
#include <stack>
#include <string>
#include <map>
#include <ctime>
#ifdef _WIN64
#include <process.h>
#include <windows.h>
#else
#include <pthread.h>
#endif
#include "cm.h"
#include "atof.h"
#include "itoa.h"
#include "compress.cu"
#include "/home/liuexp/Downloads/cudpp_src_2.0/include/cudpp_hash.h"


#ifdef _WIN64
#define fseeko _fseeki64
#define ftello _ftelli64
#else
#define _FILE_OFFSET_BITS 64
#define fseeko fseek
#define ftello ftell
#endif


using namespace std;
using namespace thrust::placeholders;

unsigned int process_count;
long long int runningRecs = 0;
long long int totalRecs = 0;
bool fact_file_loaded = 0;
bool buffersEmpty = 0;
unsigned long long int total_count = 0;
unsigned int total_segments = 0;
unsigned int total_max;
void* d_v = NULL;
void* s_v = NULL;
unsigned int curr_segment = 0;


map<string,queue<string> > top_type;
map<string,queue<string> > top_value;
map<string,queue<int_type> > top_nums;
map<string,queue<float_type> > top_nums_f;


template <typename HeadFlagType>
struct head_flag_predicate
        : public thrust::binary_function<HeadFlagType,HeadFlagType,bool>
{
    __host__ __device__
    bool operator()(HeadFlagType left, HeadFlagType right) const
    {
        return !left;
    }
};

struct is_match
{
    __host__ __device__
    bool operator()(unsigned int x)
    {
        return x != 4294967295;
    }
};

struct f_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((x-y) < EPSILON) && ((x-y) > -EPSILON));
    }
};


struct f_less
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((y-x) > EPSILON);
    }
};

struct f_greater
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((x-y) > EPSILON);
    }
};

struct f_greater_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((x-y) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
    }
};

struct f_less_equal
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((y-x) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
    }
};

struct f_not_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return !(((x-y) < EPSILON) && ((x-y) > -EPSILON));
    }
};


struct long_to_float_type
{
    __host__ __device__
    float_type operator()(const int_type x)
    {
        return (float_type)x;
    }
};


struct l_to_ui
{
    __host__ __device__
    float_type operator()(const int_type x)
    {
        return (unsigned int)x;
    }
};

struct float_to_decimal
{
    __host__ __device__
    float_type operator()(const float_type x)
    {
        return (int_type)(x*100);
    }
};


struct to_zero
{
    __host__ __device__
    bool operator()(const int_type x)
    {
        if(x == -1)
            return 0;
        else
            return 1;
    }
};



struct div_long_to_float_type
{
    __host__ __device__
    float_type operator()(const int_type x, const float_type y)
    {
        return (float_type)x/y;
    }
};

struct float_to_long
{

    __host__ __device__
    long long int operator()(const float_type x)
    {
        if ((long long int)((x+EPSILON)*100.0) > (long long int)(x*100.0))
            return (long long int)((x+EPSILON)*100.0);
        else return (long long int)(x*100.0);


    }
};

struct long_to_float
{
    __host__ __device__
    float_type operator()(const long long int x)
    {
        return (((float_type)x)/100.0);
    }
};

struct Uint2Sum
{
	__host__ __device__  uint2 operator()(uint2& a, uint2& b)
	{
		//a.x += b.x;
		a.y += b.y;
		return a;
	}	
};


struct uint2_split
{

    const uint2* d_res;
    unsigned int * output;

    uint2_split(const uint2* _d_res, unsigned int * _output):
        d_res(_d_res), output(_output) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        output[i] = d_res[i].y;
		
    }
};




struct join_functor
{

    const uint2* d_res;
	const unsigned int* d_addr;
    unsigned int * output;
    unsigned int * output1;	

    join_functor(const uint2* _d_res, const unsigned int * _d_addr, unsigned int * _output, unsigned int * _output1):
        d_res(_d_res), d_addr(_d_addr), output(_output), output1(_output1) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

	    if (d_res[i].x || d_res[i].y) {
		    for(unsigned int z = 0; z < d_res[i].y; z++) {
                output[d_addr[i] + z] = i;
                output1[d_addr[i] + z] = d_res[i].x + z;
			};	
		};		
    }
};



struct cmp_functor
{
    const char * src;
    int_type * output;
    const char * str;
    const unsigned int * len;

    cmp_functor(const char * _src, int_type * _output, const char * _str, const unsigned int * _len):
        src(_src), output(_output), str(_str), len(_len) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        if(src[i] != 0 && output[i] >= 0 && output[i] < *len ) {
            if ( src[i] == str[(*len-output[i]) - 1])
                output[i]++;
            else
                output[i] = -1;
        };
    }
};

class CudaSet;
void LoadBuffers(void* file_name);
void* LoadBuffers1(void* file_name);
void allocColumns(CudaSet* a, queue<string> fields);
void gatherColumns(CudaSet* a, CudaSet* t, string field, unsigned int segment);
void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment);
void copyGatherJoin(CudaSet* a, thrust::device_ptr<unsigned int>& m, string field, unsigned int segment, unsigned int& cnt);

map<string,CudaSet*> varNames; //  STL map to manage CudaSet variables
map<string,string> setMap; //map to keep track of column names and set names

CudaSet* bck;
queue<string> bck_fields;
bool all_copied = 0;
unsigned int bck_segment;

unsigned int findSegmentCount(char* file_name);
CudaSet *th;
bool buffersLoaded;

size_t getFreeMem();
bool zone_map_check(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums,queue<float_type> op_nums_f, CudaSet* a);


class CudaChar
{
public:
    std::vector<thrust::host_vector<char> > h_columns;
    std::vector<thrust::device_vector<char> > d_columns;
    thrust::host_vector<char> compressed;
    unsigned int mColumnCount;
    unsigned int mRecCount;


    CudaChar(unsigned int columnCount, unsigned int Recs)
        : mColumnCount(0),
          mRecCount(0)
    {
        initialize(columnCount, Recs);
    }

    CudaChar(unsigned int columnCount, unsigned int Recs, bool gpu)
        : mColumnCount(0),
          mRecCount(0)
    {
        initialize(columnCount, Recs, gpu);
    }

    CudaChar(unsigned int columnCount, unsigned int Recs, bool gpu, long long int compressed_size)
        : mColumnCount(0),
          mRecCount(0)
    {
        initialize(columnCount, Recs, gpu, compressed_size);
    }


    void findMinMax(string& minStr, string& maxStr)
    {
        thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(mRecCount);
        thrust::sequence(permutation, permutation+mRecCount);

        unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);
        void* temp;
        CUDA_SAFE_CALL(hipMalloc((void **) &temp, mRecCount));

        for(int j=mColumnCount-1; j>=0 ; j--)
            update_permutation(d_columns[j], raw_ptr, mRecCount, "ASC", (char*)temp);

        minStr = "";
        maxStr = "";

        for(unsigned int j=0; j<mColumnCount; j++) {
            minStr+=(d_columns[j])[permutation[0]];
            maxStr+=(d_columns[j])[permutation[mRecCount-1]];
        };

        hipFree(temp);
        hipFree(raw_ptr);
    }


    void resize(unsigned int addRecs)
    {
        mRecCount = mRecCount + addRecs;
        for(unsigned int i=0; i <mColumnCount; i++)
            h_columns[i].resize(mRecCount);
    }

    void allocOnDevice(unsigned int RecordCount)
    {
        mRecCount = RecordCount;
        for(unsigned int i=0; i <mColumnCount; i++)
            d_columns[i].resize(mRecCount);

    }

    void deAllocOnDevice()
    {
        if (d_columns.size())
            for(unsigned int i=0; i <mColumnCount; i++) {
                d_columns[i].resize(0);
                d_columns[i].shrink_to_fit();
            };
    };


    void CopyToGpu(unsigned int offset, unsigned int count)
    {
        for(unsigned int i = 0; i < mColumnCount; i++)
            thrust::copy(h_columns[i].begin() + offset, h_columns[i].begin() + offset +count, d_columns[i].begin());
    };


    void CopyToHost(unsigned int offset, unsigned int count)
    {
        for(unsigned int i = 0; i < mColumnCount; i++)
            thrust::copy(d_columns[i].begin(), d_columns[i].begin() + count, h_columns[i].begin() + offset);
    };


    bool* cmpStr(string str)
    {

        if (str[str.size()-1] == '%' && str[0] == '%') { // contains
            if(str.size() > mColumnCount) {
                thrust::device_ptr<bool> res_f = thrust::device_malloc<bool>(mRecCount);
                thrust::sequence(res_f, res_f+mRecCount, 0, 0);
                return thrust::raw_pointer_cast(res_f);
            }
            else {

                return 0;

            };
        }
        else if(str[str.size()-1] == '%') {  // startsWith

            if(str.size() > mColumnCount) {
                thrust::device_ptr<bool> res_f = thrust::device_malloc<bool>(mRecCount);
                thrust::sequence(res_f, res_f+mRecCount, 0, 0);
                return thrust::raw_pointer_cast(res_f);
            }
            else {

                thrust::device_ptr<bool> v = thrust::device_malloc<bool>(mRecCount);

                str.erase(str.size()-1,1);
                thrust::device_ptr<bool> res = thrust::device_malloc<bool>(mRecCount);
                thrust::sequence(res, res+mRecCount, 1, 0);

                for(int i = 0; i < str.size()-1; i++) {
                    thrust::transform(d_columns[i].begin(), d_columns[i].begin()+mRecCount, thrust::constant_iterator<char>(str[i]), v, thrust::equal_to<char>());
                    thrust::transform(v, v+mRecCount, res, res, thrust::logical_and<bool>());
                };
                thrust::device_free(v);
                return thrust::raw_pointer_cast(res);
            };

        }
        else if(str[0] == '%' ) {  // endsWith

            str.erase(0,1);
            thrust::device_ptr<char> dev_str = thrust::device_malloc<char>(str.size());
            thrust::device_ptr<unsigned int> len = thrust::device_malloc<unsigned int>(1);
            thrust::device_ptr<int_type> output = thrust::device_malloc<int_type>(mRecCount);
            thrust::device_ptr<bool> res = thrust::device_malloc<bool>(mRecCount);
            thrust::sequence(output, output+mRecCount, 0, 0);

            len[0] = str.size();
            for(int z=0; z < str.size(); z++)
                dev_str[z] = str[z];

            for(int i = mColumnCount-1; i >= 0; i--) {
                thrust::counting_iterator<unsigned int, thrust::device_space_tag> begin(0);
                cmp_functor ff(thrust::raw_pointer_cast(d_columns[i].data()),
                               thrust::raw_pointer_cast(output),
                               thrust::raw_pointer_cast(dev_str),
                               thrust::raw_pointer_cast(len));
                thrust::for_each(begin, begin + mRecCount, ff);
            };
            thrust::transform(output, output+mRecCount, res, to_zero());
            return thrust::raw_pointer_cast(res);
        }
        else {                          // equal
            thrust::device_ptr<bool> v = thrust::device_malloc<bool>(mRecCount);
            thrust::device_ptr<bool> res = thrust::device_malloc<bool>(mRecCount);
            thrust::sequence(res, res+mRecCount, 1, 0);

            if(mColumnCount < str.length())
            {
                thrust::sequence(res, res+mRecCount, 0, 0);
                return thrust::raw_pointer_cast(res);
            };

            for(unsigned int i = 0; i < mColumnCount; i++) {
                if (str.length() >= i+1)
                    thrust::transform(d_columns[i].begin(), d_columns[i].begin()+mRecCount, thrust::constant_iterator<char>(str[i]), v, thrust::equal_to<char>());
                else
                    thrust::transform(d_columns[i].begin(), d_columns[i].begin()+mRecCount, thrust::constant_iterator<char>(0), v, thrust::equal_to<char>());
                thrust::transform(v, v+mRecCount, res, res, thrust::logical_and<int_type>());
            };
            thrust::device_free(v);
            return thrust::raw_pointer_cast(res);
        };
    };


protected: // methods

    void initialize(unsigned int columnCount, unsigned int Recs)
    {
        mColumnCount = columnCount;
        mRecCount = Recs;

        for(unsigned int i=0; i <mColumnCount; i++) {
            h_columns.push_back(thrust::host_vector<char>(Recs));
            d_columns.push_back(thrust::device_vector<char>());
        };
    };

    void initialize(unsigned int columnCount, unsigned int Recs, bool gpu)
    {
        mColumnCount = columnCount;
        mRecCount = Recs;

        for(unsigned int i=0; i <mColumnCount; i++) {
            h_columns.push_back(thrust::host_vector<char>());
            d_columns.push_back(thrust::device_vector<char>());
        };
    };

    void initialize(unsigned int columnCount, unsigned int Recs, bool gpu, long long int compressed_size)
    {
        mColumnCount = columnCount;
        mRecCount = Recs;

        for(unsigned int i=0; i <mColumnCount; i++) {
            h_columns.push_back(thrust::host_vector<char>());
            d_columns.push_back(thrust::device_vector<char>());
        };
        compressed.resize(compressed_size);
    };


};



class CudaSet
{
public:
    std::vector<thrust::host_vector<int_type> > h_columns_int;
    std::vector<thrust::host_vector<float_type> > h_columns_float;
    std::vector<thrust::host_vector<char> > h_columns_char;
    std::vector<CudaChar*> h_columns_cuda_char;

    std::vector<thrust::device_vector<int_type> > d_columns_int;
    std::vector<thrust::device_vector<float_type> > d_columns_float;
    thrust::device_vector<unsigned int> prm_d;
    map<string, std::vector<unsigned int*> > prm; //represents an op's permutation of original data vectors
    //string is a set name
    //unsigned int* is an adress of the permutation array
    map<string, std::vector<unsigned int> > prm_count;	// counts of prm permutations

    map<unsigned int, unsigned int> type_index;

    unsigned int mColumnCount;
    unsigned int mRecCount;
    map<string,int> columnNames;
    map<string, FILE*> filePointers;
    bool *grp;
    queue<string> columnGroups;
    bool fact_table; // 1 = host recs are not compressed, 0 = compressed
    FILE *file_p;
    unsigned long long int *offsets; // to store the current offsets for compression routines
    unsigned int *seq;
    bool keep;
    unsigned int segCount, maxRecs;
    string name;
    //CudaSet* filter_ref;
    char* load_file_name;
    unsigned int oldRecCount;

    unsigned int* type; // 0 - integer, 1-float_type, 2-char
    bool* decimal; // column is decimal - affects only compression
    unsigned int* grp_type; // type of group : SUM, AVG, COUNT etc
    unsigned int* cols; // column positions in a file
    unsigned int grp_count;
    bool partial_load;
    bool isJoined;

    CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, int_type Recs)
        : mColumnCount(0),
          mRecCount(0)
    {
        initialize(nameRef, typeRef, sizeRef, colsRef, Recs);
        keep = false;
        offsets = 0;
        partial_load = 0;
        isJoined = 0;
    }

    CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, int_type Recs, char* file_name)
        : mColumnCount(0),
          mRecCount(0)
    {
        initialize(nameRef, typeRef, sizeRef, colsRef, Recs, file_name);
        keep = false;
        offsets = 0;
        partial_load = 1;
        isJoined = 0;
    }



    CudaSet(unsigned int RecordCount, unsigned int ColumnCount)
    {
        initialize(RecordCount, ColumnCount);
        keep = false;
        offsets = 0;
        partial_load = 0;
        isJoined = 0;
    };



    CudaSet(CudaSet* a, CudaSet* b, int_type Recs, queue<string> op_sel, queue<string> op_sel_as)
    {
        initialize(a,b,Recs, op_sel, op_sel_as);
        keep = false;
        partial_load = 0;
        isJoined = 1;
    };


    ~CudaSet()
    {
        free();
    }



    void resize(unsigned int addRecs)
    {
        mRecCount = mRecCount + addRecs;

        for(unsigned int i=0; i <mColumnCount; i++) {
            if(type[i] == 0)
                h_columns_int[type_index[i]].resize(mRecCount);
            else if(type[i] == 1)
                h_columns_float[type_index[i]].resize(mRecCount);
            else
                h_columns_cuda_char[type_index[i]]->resize(addRecs);
        };
    }


    void allocColumnOnDevice(unsigned int colIndex, unsigned int RecordCount)
    {
        if (type[colIndex] == 0)
            d_columns_int[type_index[colIndex]].resize(RecordCount);
        else if (type[colIndex] == 1)
            d_columns_float[type_index[colIndex]].resize(RecordCount);
        else
            h_columns_cuda_char[type_index[colIndex]]->allocOnDevice(RecordCount);
    };


    void deAllocColumnOnDevice(unsigned int colIndex)
    {
        if (type[colIndex] == 0 && d_columns_int.size()) {
            d_columns_int[type_index[colIndex]].resize(0);
            d_columns_int[type_index[colIndex]].shrink_to_fit();
        }
        else if (type[colIndex] == 1 && d_columns_float.size()) {
            d_columns_float[type_index[colIndex]].resize(0);
            d_columns_float[type_index[colIndex]].shrink_to_fit();
        }
        else if (type[colIndex] == 2 && h_columns_cuda_char.size())
            h_columns_cuda_char[type_index[colIndex]]->deAllocOnDevice();
    };

    void setTypes(CudaSet* b)
    {
        for(unsigned int i=0; i < b->mColumnCount; i++)
            type[i] = b->type[i];

    };

    void allocOnDevice(unsigned int RecordCount)
    {
        for(unsigned int i=0; i < mColumnCount; i++)
            allocColumnOnDevice(i, RecordCount);
    };

    void deAllocOnDevice()
    {
        for(unsigned int i=0; i <mColumnCount; i++)
            deAllocColumnOnDevice(i);
        if(!columnGroups.empty() && mRecCount !=0) {
            hipFree(grp);
            grp = NULL;
        };
    };

    void resizeDeviceColumn(unsigned int RecCount, unsigned int colIndex)
    {
        if (RecCount) {
            if (type[colIndex] == 0)
                d_columns_int[type_index[colIndex]].resize(mRecCount+RecCount);
            else if (type[colIndex] == 1)
                d_columns_float[type_index[colIndex]].resize(mRecCount+RecCount);
            else {
                for(unsigned int i = 0; i < h_columns_cuda_char[type_index[colIndex]]->mColumnCount; i++)
                    (h_columns_cuda_char[type_index[colIndex]]->d_columns[i]).resize(mRecCount+RecCount);
            };
        };
    };



    void resizeDevice(unsigned int RecCount)
    {
        if (RecCount)
            for(unsigned int i=0; i < mColumnCount; i++)
                resizeDeviceColumn(RecCount, i);
    };

    bool onDevice(unsigned int i)
    {
        unsigned j = type_index[i];

        if (type[i] == 0) {
            if (!d_columns_int.size())
                return 0;
            if (d_columns_int[j].size() == 0)
                return 0;
        }
        else if (type[i] == 1) {
            if (!d_columns_float.size())
                return 0;
            if(d_columns_float[j].size() == 0)
                return 0;
        }
        else if  (type[i] == 2) {
            if(!h_columns_cuda_char.size())
                return 0;
            if(h_columns_cuda_char[j]->d_columns[0].size() == 0)
                return 0;
        };
        return 1;
    }





    CudaSet* copyStruct(unsigned int mCount)
    {

        CudaSet* a = new CudaSet(mCount, mColumnCount);
        a->fact_table = fact_table;

        for ( map<string,int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it )
            a->columnNames[(*it).first] = (*it).second;

        for(unsigned int i=0; i < mColumnCount; i++) {
            a->cols[i] = cols[i];
            a->type[i] = type[i];

            if (a->type[i] == 0) {
                a->h_columns_int.push_back(thrust::host_vector<int_type>(mCount));
                a->d_columns_int.push_back(thrust::device_vector<int_type>());
                a->type_index[i] = a->h_columns_int.size()-1;
            }
            else if (a->type[i] == 1) {
                a->h_columns_float.push_back(thrust::host_vector<float_type>(mCount));
                a->d_columns_float.push_back(thrust::device_vector<float_type>());
                a->type_index[i] = a->h_columns_float.size()-1;
            }
            else {
                a->h_columns_cuda_char.push_back(new CudaChar((h_columns_cuda_char[type_index[i]])->mColumnCount, mCount));
                a->type_index[i] = a->h_columns_cuda_char.size()-1;
            };
        };
        return a;
    }

    CudaSet* copyDeviceStruct()
    {

        CudaSet* a = new CudaSet(mRecCount, mColumnCount);
        a->fact_table = fact_table;
        a->segCount = segCount;
        a->maxRecs = 0;

        for ( map<string,int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it )
            a->columnNames[(*it).first] = (*it).second;

        for(unsigned int i=0; i < mColumnCount; i++) {
            a->cols[i] = cols[i];
            a->type[i] = type[i];

            if(a->type[i] == 0) {
                a->d_columns_int.push_back(thrust::device_vector<int_type>());
                a->h_columns_int.push_back(thrust::host_vector<int_type>());
                a->type_index[i] = a->d_columns_int.size()-1;
            }
            else if(a->type[i] == 1) {
                a->d_columns_float.push_back(thrust::device_vector<float_type>());
                a->h_columns_float.push_back(thrust::host_vector<float_type>());
                a->type_index[i] = a->d_columns_float.size()-1;
                a->decimal[i] = decimal[i];
            }
            else {
                a->h_columns_cuda_char.push_back(new CudaChar((h_columns_cuda_char[type_index[i]])->mColumnCount, mRecCount, 0));
                a->type_index[i] = a->h_columns_cuda_char.size()-1;
            };
        };

        if(!a->fact_table) {
            a->offsets = new unsigned long long int[mColumnCount];
            for(unsigned int i =0; i < mColumnCount; i++)
                a->offsets[i] = 0;
        };

        a->mRecCount = 0;
        return a;
    }



    unsigned long long int readSegmentsFromFile(unsigned int segNum, unsigned int colIndex)
    {
        char f1[100];
        strcpy(f1, load_file_name);
        strcat(f1,".");
        char col_pos[3];
        itoaa(cols[colIndex],col_pos);
        strcat(f1,col_pos);
        FILE* f;
        int cnt, grp_count;
        unsigned long long int offset = 0;

        f = fopen (f1 , "rb" );
//        cout << "file " << f1 << " " << segNum << endl;

        for(unsigned int i = 0; i < segNum; i++) {

            if(type[colIndex] != 2) {
                fread((char *)&cnt, 4, 1, f);
                offset = offset + cnt + 8;
                fseeko(f, offset*8 , SEEK_SET);
           }
            else {
                fread((char *)&cnt, 4, 1, f);
                offset = offset + cnt*8 + 12;
                fseeko(f, offset , SEEK_SET);
                fread((char *)&grp_count, 4, 1, f);
                CudaChar* c = h_columns_cuda_char[type_index[colIndex]];
                offset = offset + 11*4 + grp_count*c->mColumnCount;
                fseeko(f, offset , SEEK_SET);
            };
        };
	
        // find out how much we need to read and rewind back to the start of the segment
        if(type[colIndex] != 2) {
            fread((char *)&cnt, 4, 1, f);
            fseeko(f, -4 , SEEK_CUR);
        }
        else {
            fread((char *)&cnt, 4, 1, f);
            offset = cnt*8 + 8;
            fseeko(f, offset , SEEK_CUR);
            fread((char *)&grp_count, 4, 1, f);
            fseeko(f, -(cnt*8+16) , SEEK_CUR);
        };

        // resize the host arrays if necessary
        // and read the segment from a file

        if(type[colIndex] == 0) {

            if(h_columns_int[type_index[colIndex]].size() < cnt+9) {
                //resize(cnt+9-h_columns_int[type_index[colIndex]].size());
				h_columns_int[type_index[colIndex]].resize(cnt+9);
            };
            fread(h_columns_int[type_index[colIndex]].data(),(cnt+8)*8,1,f);

        }
        else if(type[colIndex] == 1) {
            if(h_columns_float[type_index[colIndex]].size() < cnt+9) {
                //resize(cnt+9-h_columns_int[type_index[colIndex]].size());
				h_columns_float[type_index[colIndex]].resize(cnt+9);
			};	
            fread(h_columns_float[type_index[colIndex]].data(),(cnt+8)*8,1,f);		
			
        }
        else {
            CudaChar* c = h_columns_cuda_char[type_index[colIndex]];
            if(c->compressed.size() < cnt*8 + 14*4 + grp_count*c->mColumnCount)
                c->compressed.resize(cnt*8 + 14*4 + grp_count*c->mColumnCount);
            fread(c->compressed.data(), cnt*8 + 14*4 + grp_count*c->mColumnCount,1,f);
        };
        fclose(f);
        return 0;
    }


    unsigned long long int readSegments(unsigned int segNum, unsigned int colIndex) // read segNum number of segments and return the offset of the next segment
    {
        unsigned long long int offset = 0; // offset measured in bytes if checking chars and in 8 byte integers if checking ints and decimals
        unsigned int grp_count;
        unsigned int data_len;

        for(unsigned int i = 0; i < segNum; i++) {
            if(type[colIndex] == 0) {
                data_len = ((unsigned int*)((h_columns_int[type_index[colIndex]]).data() + offset))[0];
                offset = offset + data_len + 8;
            }
            else if(type[colIndex] == 1) {
                data_len = ((unsigned int*)((h_columns_float[type_index[colIndex]]).data() + offset))[0];
                offset = offset + data_len + 8;
            }
            else {
                CudaChar* c = h_columns_cuda_char[type_index[colIndex]];
                data_len = ((unsigned int*)(c->compressed.data() + offset))[0];
                grp_count = ((unsigned int*)(c->compressed.data() + offset + 8*data_len + 12))[0];
                offset = offset + data_len*8 + 14*4 + grp_count*c->mColumnCount;
            };
        };
        return offset;
    }


    void CopyToGpu(unsigned int offset, unsigned int count)
    {
        if (fact_table) {
            for(unsigned int i = 0; i < mColumnCount; i++) {
                switch(type[i]) {
                case 0 :
                    thrust::copy(h_columns_int[type_index[i]].begin() + offset, h_columns_int[type_index[i]].begin() + offset + count, d_columns_int[type_index[i]].begin());
                    break;
                case 1 :
                    thrust::copy(h_columns_float[type_index[i]].begin() + offset, h_columns_float[type_index[i]].begin() + offset + count, d_columns_float[type_index[i]].begin());
                    break;
                default :
                    (h_columns_cuda_char[type_index[i]])->CopyToGpu(offset, count);
                };
            };
        }
        else
            for(unsigned int i = 0; i < mColumnCount; i++)
                CopyColumnToGpu(i,  offset, count);
    }




    void CopyToGpu(unsigned int segment)
    {
        if (fact_table) {
            for(unsigned int i = 0; i < mColumnCount; i++) {
                switch(type[i]) {
                case 0 :
                    thrust::copy(h_columns_int[type_index[i]].begin(), h_columns_int[type_index[i]].begin() + mRecCount, d_columns_int[type_index[i]].begin());
                    break;
                case 1 :
                    thrust::copy(h_columns_float[type_index[i]].begin(), h_columns_float[type_index[i]].begin() + mRecCount, d_columns_float[type_index[i]].begin());
                    break;
                default :
                    (h_columns_cuda_char[type_index[i]])->CopyToGpu(0, mRecCount);
                };
            };
        }
        else
            for(unsigned int i = 0; i < mColumnCount; i++)
                CopyColumnToGpu(i,  segment);
    }





    void CopyColumnToGpu(unsigned int colIndex,  unsigned int segment)
    {
        if(fact_table) {
            switch(type[colIndex]) {
            case 0 :
                thrust::copy(h_columns_int[type_index[colIndex]].begin(), h_columns_int[type_index[colIndex]].begin() + mRecCount, d_columns_int[type_index[colIndex]].begin());
                break;
            case 1 :
                thrust::copy(h_columns_float[type_index[colIndex]].begin(), h_columns_float[type_index[colIndex]].begin() + mRecCount, d_columns_float[type_index[colIndex]].begin());
                break;
            default :
                (h_columns_cuda_char[type_index[colIndex]])->CopyToGpu(0, mRecCount);
            };
        }
        else {
            //cout << "start " << colIndex << " " << type[colIndex] << " " << segment << " " << partial_load << endl;
            unsigned long long int data_offset;
            if (partial_load)
                data_offset = readSegmentsFromFile(segment,colIndex);
            else
                data_offset = readSegments(segment,colIndex);

				
            if(d_v == NULL)
                CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
            if(s_v == NULL);
                CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));
			

            switch(type[colIndex]) {
            case 0 :
                pfor_decompress(thrust::raw_pointer_cast(d_columns_int[type_index[colIndex]].data()), h_columns_int[type_index[colIndex]].data() + data_offset, &mRecCount, 0, NULL, d_v, s_v);
                break;
            case 1 :
                if(decimal[colIndex]) {
                    pfor_decompress( thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data()) , h_columns_float[type_index[colIndex]].data() + data_offset, &mRecCount, 0, NULL, d_v, s_v);
                    thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data()));
                    thrust::transform(d_col_int,d_col_int+mRecCount,d_columns_float[type_index[colIndex]].begin(), long_to_float());
                }
                //else // uncompressed float
                //hipMemcpy( d_columns[colIndex], (void *) ((float_type*)h_columns[colIndex] + offset), count*float_size, hipMemcpyHostToDevice);
                // will have to fix it later so uncompressed data will be written by segments too
                break;
            default :
                CudaChar* c = h_columns_cuda_char[type_index[colIndex]];
                unsigned int data_len = ((unsigned int*)(c->compressed.data() + data_offset))[0];
                grp_count = ((unsigned int*)(c->compressed.data() + data_offset + data_len*8 + 12))[0];
                pfor_dict_decompress(c->compressed.data() + data_offset, c->h_columns , c->d_columns, &mRecCount, NULL,0, c->mColumnCount, 0, d_v, s_v);
            };
            //hipFree(d_v);
            //hipFree(s_v);
        };
    }



    void CopyColumnToGpu(unsigned int colIndex) // copy all segments
    {
        if(fact_table) {
            switch(type[colIndex]) {
            case 0 :
                thrust::copy(h_columns_int[type_index[colIndex]].begin(), h_columns_int[type_index[colIndex]].begin() + mRecCount, d_columns_int[type_index[colIndex]].begin());
                break;
            case 1 :
                thrust::copy(h_columns_float[type_index[colIndex]].begin(), h_columns_float[type_index[colIndex]].begin() + mRecCount, d_columns_float[type_index[colIndex]].begin());
                break;
            default :
                (h_columns_cuda_char[type_index[colIndex]])->CopyToGpu(0, mRecCount);
            };
        }
        else {
            long long int data_offset;
            unsigned int totalRecs = 0;
            if(d_v == NULL)
                CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
            if(s_v == NULL);
                CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));

            for(unsigned int i = 0; i < segCount; i++) {

                if (partial_load)
                    data_offset = readSegmentsFromFile(i,colIndex);
                else
                    data_offset = readSegments(i,colIndex);
                switch(type[colIndex]) {
                case 0 :
                    pfor_decompress(thrust::raw_pointer_cast(d_columns_int[type_index[colIndex]].data() + totalRecs), h_columns_int[type_index[colIndex]].data() + data_offset, &mRecCount, 0, NULL, d_v, s_v);
                    break;
                case 1 :
                    if(decimal[colIndex]) {
                        pfor_decompress( thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data() + totalRecs) , h_columns_float[type_index[colIndex]].data() + data_offset, &mRecCount, 0, NULL, d_v, s_v);
                        thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data() + totalRecs));
                        thrust::transform(d_col_int,d_col_int+mRecCount,d_columns_float[type_index[colIndex]].begin() + totalRecs, long_to_float());
                    }
                    // else  uncompressed float
                    //hipMemcpy( d_columns[colIndex], (void *) ((float_type*)h_columns[colIndex] + offset), count*float_size, hipMemcpyHostToDevice);
                    // will have to fix it later so uncompressed data will be written by segments too
                    break;
                default :
                    CudaChar* c = h_columns_cuda_char[type_index[colIndex]];
                    pfor_dict_decompress(c->compressed.data() + data_offset, c->h_columns , c->d_columns, &mRecCount, NULL,0, c->mColumnCount, totalRecs, d_v, s_v);
                };
                totalRecs = totalRecs + mRecCount;
            };
            hipFree(d_v);
            hipFree(s_v);

            mRecCount = totalRecs;
        };
    }




    void CopyColumnToGpu(unsigned int colIndex,  unsigned int offset, unsigned int count)
    {
        if(fact_table) {
            switch(type[colIndex]) {
            case 0 :
                thrust::copy(h_columns_int[type_index[colIndex]].begin(), h_columns_int[type_index[colIndex]].begin() + offset + count, d_columns_int[type_index[colIndex]].begin());
                break;
            case 1 :
                thrust::copy(h_columns_float[type_index[colIndex]].begin(), h_columns_float[type_index[colIndex]].begin() + offset + count, d_columns_float[type_index[colIndex]].begin());
                break;
            default :
                (h_columns_cuda_char[type_index[colIndex]])->CopyToGpu(offset, count);
            };
        }
        else {
            unsigned int start_seg, seg_num, grp_count, data_len, mCount;
            start_seg = offset/segCount; // starting segment
            seg_num = count/segCount;    // number of segments that we need
            long long int data_offset;
            if(partial_load)
                data_offset = readSegmentsFromFile(start_seg,colIndex);
            else
                data_offset = readSegments(start_seg,colIndex);

            if(d_v == NULL)
                CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
            if(s_v == NULL);
                CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));

            switch(type[colIndex]) {
            case 0 :
                for(unsigned int j = 0; j < seg_num; j++) {
                    data_len = ((unsigned int*)(h_columns_int[type_index[colIndex]].data()))[data_offset];
                    pfor_decompress(thrust::raw_pointer_cast(d_columns_int[type_index[colIndex]].data() + segCount*j), h_columns_int[type_index[colIndex]].data() + data_offset, &data_len, 0, NULL, d_v, s_v);
                    data_offset = data_offset + data_len + 8;
                };
                break;
            case 1 :
                if(decimal[colIndex]) {
                    for(unsigned int j = 0; j < seg_num; j++) {
                        data_len = (((unsigned int*)(h_columns_int[type_index[colIndex]]).data()))[data_offset];
                        thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data() + segCount*j));
                        pfor_decompress( thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data() + segCount*j), h_columns_float[type_index[colIndex]].data() + data_offset, &data_len, 0, NULL, d_v, s_v);
                        thrust::transform(d_col_int,d_col_int+mRecCount,d_columns_float[type_index[colIndex]].begin() + segCount*j, long_to_float());
                        data_offset = data_offset + data_len + 8;
                    };
                }
                else // uncompressed float
                    thrust::copy(h_columns_float[type_index[colIndex]].begin() + offset, h_columns_float[type_index[colIndex]].begin() + offset + count, d_columns_float[type_index[colIndex]].begin());
                break;
            default :
                CudaChar* c = h_columns_cuda_char[type_index[colIndex]];
                for(unsigned int j = 0; j < seg_num; j++) {
                    data_len = ((unsigned int*)(c->compressed.data() + data_offset))[0];
                    grp_count = ((unsigned int*)(c->compressed.data() + data_offset + data_len*8 + 12))[0];
                    pfor_dict_decompress(c->compressed.data() + data_offset, c->h_columns , c->d_columns, &mCount, NULL,0, c->mColumnCount, segCount*j, d_v, s_v);
                    data_offset = data_offset + data_len*8 + 14*4 + grp_count*c->mColumnCount;
                };
            };
            hipFree(d_v);
            hipFree(s_v);
        };
    }



    void CopyColumnToHost(int colIndex, unsigned int offset, unsigned int RecCount)
    {
        if(fact_table) {
            switch(type[colIndex]) {
            case 0 :
                thrust::copy(d_columns_int[type_index[colIndex]].begin(), d_columns_int[type_index[colIndex]].begin() + RecCount, h_columns_int[type_index[colIndex]].begin() + offset);
                break;
            case 1 :
                thrust::copy(d_columns_float[type_index[colIndex]].begin(), d_columns_float[type_index[colIndex]].begin() + RecCount, h_columns_float[type_index[colIndex]].begin() + offset);
                break;
            default :
                (h_columns_cuda_char[type_index[colIndex]])->CopyToHost(offset,RecCount);
            }
        }
        else {
            unsigned long long int comp_offset = 0;
            switch(type[colIndex]) {
            case 0 :
                comp_offset = pfor_compress(thrust::raw_pointer_cast(d_columns_int[type_index[colIndex]].data()), RecCount*int_size, NULL, h_columns_int[type_index[colIndex]], 0, comp_offset);
                break;
            case 1 :
                if (decimal[colIndex]) {
                    thrust::device_ptr<long long int> d_col_dec((long long int*)thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data() ));
                    thrust::transform(d_columns_float[type_index[colIndex]].begin(), d_columns_float[type_index[colIndex]].begin()+RecCount,
                                      d_col_dec, float_to_long());
                    comp_offset = pfor_compress(thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data()), RecCount*float_size, NULL, h_columns_float[type_index[colIndex]], 0, comp_offset);
                }
                else { // add code for float
                } ;
                break;
            default :
                CudaChar *s = (h_columns_cuda_char)[type_index[colIndex]];
                comp_offset = pfor_dict_compress(s->d_columns, s->mColumnCount, NULL, RecCount, s->compressed, comp_offset);
            };
        };
    }


    void CopyColumnToHost(int colIndex)
    {
        CopyColumnToHost(colIndex, 0, mRecCount);
    }

    void CopyToHost(unsigned int offset, unsigned int count)
    {
        for(unsigned int i = 0; i < mColumnCount; i++)
            CopyColumnToHost(i, offset, count);
    }

    float_type* get_float_type_by_name(string name)
    {
        unsigned int colIndex = columnNames.find(name)->second;
        return thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data());
    }

    int_type* get_int_by_name(string name)
    {
        unsigned int colIndex = columnNames.find(name)->second;
        return thrust::raw_pointer_cast(d_columns_int[type_index[colIndex]].data());
    }

    float_type* get_host_float_by_name(string name)
    {
        unsigned int colIndex = columnNames.find(name)->second;
        return thrust::raw_pointer_cast(h_columns_float[type_index[colIndex]].data());
    }

    int_type* get_host_int_by_name(string name)
    {
        unsigned int colIndex = columnNames.find(name)->second;
        return thrust::raw_pointer_cast(h_columns_int[type_index[colIndex]].data());
    }



    void GroupBy(queue<string> columnRef)
    {
        int grpInd, colIndex;

        if(!columnGroups.empty())
            hipFree(grp);

        CUDA_SAFE_CALL(hipMalloc((void **) &grp, mRecCount * sizeof(bool))); // d_di is the vector for segmented scans
        thrust::device_ptr<bool> d_grp(grp);

        thrust::sequence(d_grp, d_grp+mRecCount, 0, 0);

        thrust::device_ptr<bool> d_group = thrust::device_malloc<bool>(mRecCount);
        d_group[mRecCount-1] = 1;

        for(int i = 0; i < columnRef.size(); columnRef.pop()) {
            columnGroups.push(columnRef.front()); // save for future references
            colIndex = columnNames[columnRef.front()];

            if(!onDevice(colIndex)) {
                allocColumnOnDevice(colIndex,mRecCount);
                CopyColumnToGpu(colIndex,  0, mRecCount);
                grpInd = 1;
            }
            else
                grpInd = 0;

            if (type[colIndex] == 0) {  // int_type
                thrust::transform(d_columns_int[type_index[colIndex]].begin(), d_columns_int[type_index[colIndex]].begin() + mRecCount - 1,
                                  d_columns_int[type_index[colIndex]].begin()+1, d_group, thrust::not_equal_to<int_type>());
                thrust::transform(d_group, d_group+mRecCount, d_grp, d_grp, thrust::logical_or<bool>());
            }
            else if (type[colIndex] == 1) {  // float_type
                thrust::transform(d_columns_float[type_index[colIndex]].begin(), d_columns_float[type_index[colIndex]].begin() + mRecCount - 1,
                                  d_columns_float[type_index[colIndex]].begin()+1, d_group, f_not_equal_to());
                thrust::transform(d_group, d_group+mRecCount, d_grp, d_grp, thrust::logical_or<bool>());
            }
            else  {  // CudaChar
                CudaChar* c = h_columns_cuda_char[type_index[colIndex]];
                for(unsigned int j=0; j < c->mColumnCount; j++) {
                    thrust::transform(c->d_columns[j].begin(), c->d_columns[j].begin() + mRecCount - 1, c->d_columns[j].begin()+1, d_group, thrust::not_equal_to<char>());
                    thrust::transform(d_group, d_group+mRecCount, d_grp, d_grp, thrust::logical_or<int>());
                }
            };
            if (grpInd == 1)
                deAllocColumnOnDevice(colIndex);
        };

        thrust::device_free(d_group);
        grp_count = thrust::count(d_grp, d_grp+mRecCount,1);
    }


    void addDeviceColumn(int_type* col, int colIndex, string colName, int_type recCount)
    {
        if (columnNames.find(colName) == columnNames.end()) {
            columnNames[colName] = colIndex;
            type[colIndex] = 0;
            d_columns_int.push_back(thrust::device_vector<int_type>(recCount));
            type_index[colIndex] = d_columns_int.size()-1;
        }
        else {  // already exists, my need to resize it
            if(mRecCount < recCount)
                resizeDeviceColumn(colIndex, recCount-mRecCount);
        };
        // copy data to d columns
        thrust::device_ptr<int_type> d_col((int_type*)col);
        thrust::copy(d_col, d_col+recCount, d_columns_int[type_index[colIndex]].begin());
        mRecCount = recCount;
    };

    void addDeviceColumn(float_type* col, int colIndex, string colName, int_type recCount)
    {
        if (columnNames.find(colName) == columnNames.end()) {
            columnNames[colName] = colIndex;
            type[colIndex] = 1;
            d_columns_float.push_back(thrust::device_vector<float_type>(recCount));
            type_index[colIndex] = d_columns_float.size()-1;
        }
        else {  // already exists, my need to resize it
            if(mRecCount < recCount)
                resizeDeviceColumn(colIndex, recCount-mRecCount);
        };

        thrust::device_ptr<float_type> d_col((float_type*)col);
        thrust::copy(d_col, d_col+recCount, d_columns_float[type_index[colIndex]].begin());
        mRecCount = recCount;
    };



    void addHostColumn(int_type* col, int colIndex, string colName, int_type recCount, int_type old_reccount, bool one_line)
    {
        if (columnNames.find(colName) == columnNames.end()) {
            columnNames[colName] = colIndex;
            type[colIndex] = 0;
            if (!one_line) {
                h_columns_int.push_back(thrust::host_vector<int_type>(old_reccount));
                type_index[colIndex] = h_columns_int.size()-1;
            }
            else {
                h_columns_int.push_back(thrust::host_vector<int_type>(1));
                type_index[colIndex] = h_columns_int.size()-1;
            };
        };

        if (!one_line) {
            thrust::device_ptr<int_type> d_col((int_type*)col);
            thrust::copy(d_col, d_col+recCount, h_columns_int[type_index[colIndex]].begin() + mRecCount);
        }
        else {
            thrust::device_ptr<int_type> src(col);
            (h_columns_int[type_index[colIndex]])[0] = (h_columns_int[type_index[colIndex]])[0] + src[0];
        };
    };

    void addHostColumn(float_type* col, int colIndex, string colName, int_type recCount, int_type old_reccount, bool one_line)
    {
        if (columnNames.find(colName) == columnNames.end()) {
            columnNames[colName] = colIndex;
            type[colIndex] = 1;
            if (!one_line) {
                h_columns_float.push_back(thrust::host_vector<float_type>(old_reccount));
                type_index[colIndex] = h_columns_float.size()-1;
            }
            else {
                h_columns_float.push_back(thrust::host_vector<float_type>(1));
                type_index[colIndex] = h_columns_float.size()-1;
            };
        };

        if (!one_line) {
            thrust::device_ptr<float_type> d_col((float_type*)col);
            thrust::copy(d_col, d_col+recCount, h_columns_float[type_index[colIndex]].begin() + mRecCount);
        }
        else {
            thrust::device_ptr<float_type> src(col);
            (h_columns_float[type_index[colIndex]])[0] = (h_columns_float[type_index[colIndex]])[0] + src[0];
        };
    };



    void Store(char* file_name, char* sep, unsigned int limit, bool binary )
    {
        if (mRecCount == 0 && binary == 1) { // write tails

            char str[100];
            char col_pos[3];

            for(unsigned int i = 0; i< mColumnCount; i++) {
                strcpy(str, file_name);
                strcat(str,".");
                itoaa(cols[i],col_pos);
                strcat(str,col_pos);

                fstream binary_file(str,ios::out|ios::binary|ios::app);
                binary_file.write((char *)&total_count, 8);
                binary_file.write((char *)&total_segments, 4);
                binary_file.write((char *)&total_max, 4);
                binary_file.close();
            };
            return;
        };



        unsigned int mCount;

        if(limit != 0 && limit < mRecCount)
            mCount = limit;
        else
            mCount = mRecCount;

        if(binary == 0) {

            if(prm.size() > 0) { // data permuted
                // allocate on device and gather
                queue<string> op_vx;
                for ( map<string,int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it )
                    op_vx.push((*it).first);

                allocColumns(this, op_vx);
                copyColumns(this, op_vx, 0);
            };

            FILE *file_pr = fopen(file_name, "w");
            if (file_pr  == NULL)
                cout << "Could not open file " << file_name << endl;

            char buffer [33];
            if(onDevice(0)) {

                if(h_columns_int.size() == 0 && h_columns_float.size() == 0) {
                    for(unsigned int i = 0; i< mColumnCount; i++)
                        if(type[i] == 0)
                            h_columns_int.push_back(thrust::host_vector<int_type>(mCount));
                        else if(type[i] == 1)
                            h_columns_float.push_back(thrust::host_vector<float_type>(mCount));
                };

                resize(mCount+1);
                bool ch = 0;
                if(!fact_table) {
                    fact_table = 1;
                    ch = 1;
                };
                CopyToHost(0,mCount);
                if(ch)
                    fact_table = 0;
            }
            else {
                if(!fact_table) { // compressed on the host
                    allocOnDevice(mCount);
                    for(unsigned int i=0; i < mColumnCount; i++) {
                        CopyColumnToGpu(i);
                        resize(mCount+1);
                    };
                    fact_table = 1;
                    CopyToHost(0,mCount);
                    fact_table = 0;
                };
            };

            for(unsigned int i=0; i < mCount; i++) {
                for(unsigned int j=0; j < mColumnCount; j++) {
                    if (type[j] == 0) {
                        sprintf(buffer, "%lld", (h_columns_int[type_index[j]])[i] );
                        fputs(buffer,file_pr);
                        fputs(sep, file_pr);
                    }
                    else if (type[j] == 1) {
                        sprintf(buffer, "%.2f", (h_columns_float[type_index[j]])[i] );
                        fputs(buffer,file_pr);
                        fputs(sep, file_pr);
                    }
                    else {
                        CudaChar* cc = h_columns_cuda_char[type_index[j]];
                        char *buf = new char[(cc->mColumnCount)+1];
                        for(unsigned int z=0; z<(cc->mColumnCount); z++)
                            buf[z] = (cc->h_columns[z])[i];
                        buf[cc->mColumnCount] = 0;
                        fputs(buf, file_pr);
                        fputs(sep, file_pr);
                        delete [] buf;
                    };
                };
                if (i != mCount -1)
                    fputs("\n",file_pr);
            };
            fclose(file_pr);
        }
        else {  //writing a binary file

            char str[100];
            char col_pos[3];
            total_count = total_count + mCount;
            total_segments = total_segments + 1;
            if (mCount > total_max)
                total_max = mCount;

            bool in_gpu = false;
            if(onDevice(0))
                in_gpu = true;

            void* d;
            if(!in_gpu)
                CUDA_SAFE_CALL(hipMalloc((void **) &d, mCount*float_size));

            void* host;
            hipHostMalloc(&host, float_size*mCount);

            for(unsigned int i = 0; i< mColumnCount; i++)
                if(type[i] == 2 && !in_gpu ) {
                    allocColumnOnDevice(i, mCount);
                    CopyColumnToGpu(i,  0, mCount);
                };

            for(unsigned int i = 0; i< mColumnCount; i++) {
                strcpy(str, file_name);
                strcat(str,".");
                itoaa(cols[i],col_pos);
                strcat(str,col_pos);

                if(type[i] == 0) {
                    if(!in_gpu) {
                        thrust::device_ptr<int_type> d_col((int_type*)d);
                        thrust::copy(h_columns_int[type_index[i]].begin(), h_columns_int[type_index[i]].begin() + mCount, d_col);
                        pfor_compress( d, mCount*int_size, str, h_columns_int[type_index[i]], 0, 0);
                    }
                    else
                        pfor_compress( thrust::raw_pointer_cast(d_columns_int[type_index[i]].data()), mCount*int_size, str, h_columns_int[type_index[i]], 0, 0);
                }
                else if(type[i] == 1) {
                    if(decimal[i]) {
                        if(!in_gpu) {
                            thrust::device_ptr<float_type> d_col((float_type*)d);
                            thrust::copy(h_columns_float[type_index[i]].begin(), h_columns_float[type_index[i]].begin() + mCount, d_col);
                            thrust::device_ptr<long long int> d_col_dec((long long int*)d);
                            thrust::transform(d_col,d_col+mCount,d_col_dec, float_to_long());
                            pfor_compress( d, mCount*float_size, str, h_columns_float[type_index[i]], 1, 0);
                        }
                        else {
                            thrust::device_ptr<long long int> d_col_dec((long long int*)(thrust::raw_pointer_cast(d_columns_float[type_index[i]].data()) ));
                            thrust::transform(d_columns_float[type_index[i]].begin(),d_columns_float[type_index[i]].begin()+mCount, d_col_dec, float_to_long());
                            pfor_compress( thrust::raw_pointer_cast(d_columns_float[type_index[i]].data()), mCount*float_size, str, h_columns_float[type_index[i]], 1, 0);
                        };
                    }
                    else { // do not compress
                        fstream binary_file(str,ios::out|ios::binary|fstream::app);
                        binary_file.write((char *)&mCount, 4);
                        if(in_gpu) {
                            hipMemcpy(host, thrust::raw_pointer_cast(d_columns_float[type_index[i]].data()), mCount*float_size, hipMemcpyDeviceToHost);
                            binary_file.write((char *)host,mCount*float_size);
                        }
                        else
                            binary_file.write((char *)(h_columns_float[type_index[i]].data()),mCount*float_size);
                        unsigned int comp_type = 3;
                        binary_file.write((char *)&comp_type, 4);
                        binary_file.close();
                    };
                }
                else {
                    CudaChar *a = h_columns_cuda_char[type_index[i]];
                    thrust::host_vector<char> hh(mCount*8);
                    pfor_dict_compress(a->d_columns, a->mColumnCount, str, mCount, hh, 0);
                };

                if(fact_file_loaded) {
                    fstream binary_file(str,ios::out|ios::binary|ios::app);
                    binary_file.write((char *)&total_count, 8);
                    binary_file.write((char *)&total_segments, 4);
                    binary_file.write((char *)&total_max, 4);
                    binary_file.close();
                };

            };


            for(unsigned int i = 0; i< mColumnCount; i++)
                if(type[i] == 2 && !in_gpu)
                    deAllocColumnOnDevice(i);

            if(!in_gpu)
                hipFree(d);
            hipHostFree(host);

        }
    }




    void LoadFile(char* file_name, char* sep )
    {
        unsigned int count = 0;
        char line[500];
        int l;
        char* field;
        unsigned int current_column = 1;

        FILE *file_ptr = fopen(file_name, "r");
        if (file_ptr  == NULL)
            cout << "Could not open file " << file_name << endl;

        unsigned int *seq = new unsigned int[mColumnCount];
        thrust::sequence(seq, seq+mColumnCount,0,1);
        thrust::stable_sort_by_key(cols, cols+mColumnCount, seq);


        while (fgets(line, 500, file_ptr) != NULL ) {

            current_column = 1;
            field = strtok(line,sep);

            for(unsigned int i = 0; i< mColumnCount; i++) {

                while(cols[i] > current_column) {
                    field = strtok(NULL,sep);
                    current_column++;
                };

                if (type[seq[i]] == 0) {
                    if (strchr(field,'-') == NULL) {
                        (h_columns_int[type_index[seq[i]]])[count] = atoi(field);
                    }
                    else {   // handling possible dates
                        strncpy(field+4,field+5,2);
                        strncpy(field+6,field+8,2);
                        field[8] = '\0';
                        (h_columns_int[type_index[seq[i]]])[count] = atoi(field);
                    };
                }
                else if (type[seq[i]] == 1)
                    (h_columns_float[type_index[seq[i]]])[count] = atoff(field);
                else {
                    l = strlen(field);
                    for(int j =0; j< l; j++)
                        ((h_columns_cuda_char[type_index[seq[i]]])->h_columns[j])[count] = field[j];
                    for(unsigned int j =l; j< (h_columns_cuda_char[type_index[i]])->mColumnCount; j++)
                        ((h_columns_cuda_char[type_index[seq[i]]])->h_columns[j])[count] = 0;
                };
            };
            count++;
            if (count == mRecCount) {
                mRecCount = mRecCount + process_count;
                resize(mRecCount);
            };
        };
        fclose(file_ptr);
        mRecCount = count;
    }


    int LoadBigFile(const char* file_name, const char* sep )
    {
        unsigned int count = 0;
        char line[500];
        char* field;
        unsigned int current_column = 1;
        unsigned int l;

        if (file_p == NULL)
            file_p = fopen(file_name, "r");
        if (file_p  == NULL)
            cout << "Could not open file " << file_name << endl;

        if (seq == 0) {
            seq = new unsigned int[mColumnCount];
            thrust::sequence(seq, seq+mColumnCount,0,1);
            thrust::stable_sort_by_key(cols, cols+mColumnCount, seq);
        };

        while (count < process_count && fgets(line, 500, file_p) != NULL) {

            current_column = 1;
            field = strtok(line,sep);

            for(unsigned int i = 0; i< mColumnCount; i++) {

                while(cols[i] > current_column) {
                    field = strtok(NULL,sep);
                    current_column++;
                };
                if (type[seq[i]] == 0) {
                    if (strchr(field,'-') == NULL) {
                        (h_columns_int[type_index[seq[i]]])[count] = atoi(field);
                    }
                    else {   // handling possible dates
                        strncpy(field+4,field+5,2);
                        strncpy(field+6,field+8,2);
                        field[8] = '\0';
                        (h_columns_int[type_index[seq[i]]])[count] = atoi(field);
                    };
                }
                else if (type[seq[i]] == 1)
                    (h_columns_float[type_index[seq[i]]])[count] = atoff(field);
                else {
                    l = strlen(field);
                    for(unsigned int j =0; j< l; j++)
                        ((h_columns_cuda_char[type_index[seq[i]]])->h_columns[j])[count] = field[j];
                    for(unsigned int j =l; j< (h_columns_cuda_char[type_index[seq[i]]])->mColumnCount; j++)
                        ((h_columns_cuda_char[type_index[seq[i]]])->h_columns[j])[count] = 0;
                };
            };
            count++;
        };

        mRecCount = count;

        if(count < process_count)  {
            fclose(file_p);
            return 1;
        }
        else
            return 0;
    }


    void free()  {

        if (!seq)
            delete seq;

        for(unsigned int i = 0; i < mColumnCount; i++ ) {
            if(type[i] == 2 && h_columns_cuda_char.size() > 0 && prm.size() == 0)
                delete h_columns_cuda_char[type_index[i]];
        };

        delete type;
        delete cols;

        if(!columnGroups.empty() && mRecCount !=0 && grp != NULL)
            hipFree(grp);

        for ( map<string, std::vector<unsigned int*> >::iterator it=prm.begin() ; it != prm.end(); ++it ) {
            for(unsigned int i = 0; i < prm[(*it).first].size(); i++)
                delete [] prm[(*it).first][i];
        };


    };


    bool* logical_and(bool* column1, bool* column2)
    {
        thrust::device_ptr<bool> dev_ptr1(column1);
        thrust::device_ptr<bool> dev_ptr2(column2);

        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_and<bool>());

        thrust::device_free(dev_ptr2);
        return column1;
    }


    bool* logical_or(bool* column1, bool* column2)
    {

        thrust::device_ptr<bool> dev_ptr1(column1);
        thrust::device_ptr<bool> dev_ptr2(column2);

        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_or<bool>());
        thrust::device_free(dev_ptr2);
        return column1;
    }



    bool* compare(int_type s, int_type d, int_type op_type)
    {
        bool res;

        if (op_type == 2) // >
            if(d>s) res = 1;
            else res = 0;
        else if (op_type == 1)  // <
            if(d<s) res = 1;
            else res = 0;
        else if (op_type == 6) // >=
            if(d>=s) res = 1;
            else res = 0;
        else if (op_type == 5)  // <=
            if(d<=s) res = 1;
            else res = 0;
        else if (op_type == 4)// =
            if(d==s) res = 1;
            else res = 0;
        else // !=
            if(d!=s) res = 1;
            else res = 0;

        thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
        thrust::sequence(p, p+mRecCount,res,(bool)0);

        return thrust::raw_pointer_cast(p);
    }


    bool* compare(float_type s, float_type d, int_type op_type)
    {
        bool res;

        if (op_type == 2) // >
            if ((d-s) > EPSILON) res = 1;
            else res = 0;
        else if (op_type == 1)  // <
            if ((s-d) > EPSILON) res = 1;
            else res = 0;
        else if (op_type == 6) // >=
            if (((d-s) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
            else res = 0;
        else if (op_type == 5)  // <=
            if (((s-d) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
            else res = 0;
        else if (op_type == 4)// =
            if (((d-s) < EPSILON) && ((d-s) > -EPSILON)) res = 1;
            else res = 0;
        else // !=
            if (!(((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
            else res = 0;

        thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
        thrust::sequence(p, p+mRecCount,res,(bool)0);

        return thrust::raw_pointer_cast(p);
    }


    bool* compare(int_type* column1, int_type d, int_type op_type)
    {
        thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);
        thrust::device_ptr<int_type> dev_ptr(column1);


        if (op_type == 2) // >
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater<int_type>());
        else if (op_type == 1)  // <
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less<int_type>());
        else if (op_type == 6) // >=
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater_equal<int_type>());
        else if (op_type == 5)  // <=
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less_equal<int_type>());
        else if (op_type == 4)// =
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::equal_to<int_type>());
        else // !=
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::not_equal_to<int_type>());

        return thrust::raw_pointer_cast(temp);

    }

    bool* compare(float_type* column1, float_type d, int_type op_type)
    {
        thrust::device_ptr<bool> res = thrust::device_malloc<bool>(mRecCount);
        thrust::device_ptr<float_type> dev_ptr(column1);

        if (op_type == 2) // >
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater());
        else if (op_type == 1)  // <
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less());
        else if (op_type == 6) // >=
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater_equal_to());
        else if (op_type == 5)  // <=
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less_equal());
        else if (op_type == 4)// =
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_equal_to());
        else // !=
            thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_not_equal_to());

        return thrust::raw_pointer_cast(res);
    }


    bool* compare(int_type* column1, int_type* column2, int_type op_type)
    {
        thrust::device_ptr<int_type> dev_ptr1(column1);
        thrust::device_ptr<int_type> dev_ptr2(column2);
        thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

        if (op_type == 2) // >
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater<int_type>());
        else if (op_type == 1)  // <
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less<int_type>());
        else if (op_type == 6) // >=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater_equal<int_type>());
        else if (op_type == 5)  // <=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less_equal<int_type>());
        else if (op_type == 4)// =
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::equal_to<int_type>());
        else // !=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::not_equal_to<int_type>());

        return thrust::raw_pointer_cast(temp);
    }

    bool* compare(float_type* column1, float_type* column2, int_type op_type)
    {
        thrust::device_ptr<float_type> dev_ptr1(column1);
        thrust::device_ptr<float_type> dev_ptr2(column2);
        thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

        if (op_type == 2) // >
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
        else if (op_type == 1)  // <
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
        else if (op_type == 6) // >=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
        else if (op_type == 5)  // <=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
        else if (op_type == 4)// =
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
        else // !=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

        return thrust::raw_pointer_cast(temp);

    }


    bool* compare(float_type* column1, int_type* column2, int_type op_type)
    {
        thrust::device_ptr<float_type> dev_ptr1(column1);
        thrust::device_ptr<int_type> dev_ptr(column2);
        thrust::device_ptr<float_type> dev_ptr2 = thrust::device_malloc<float_type>(mRecCount);;
        thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

        thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr2, long_to_float_type());

        if (op_type == 2) // >
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
        else if (op_type == 1)  // <
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
        else if (op_type == 6) // >=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
        else if (op_type == 5)  // <=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
        else if (op_type == 4)// =
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
        else // !=
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

        thrust::device_free(dev_ptr2);
        return thrust::raw_pointer_cast(temp);
    }


    float_type* op(int_type* column1, float_type* column2, string op_type, int reverse)
    {

        thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
        thrust::device_ptr<int_type> dev_ptr(column1);

        thrust::transform(dev_ptr, dev_ptr + mRecCount, temp, long_to_float_type()); // in-place transformation

        thrust::device_ptr<float_type> dev_ptr1(column2);

        if(reverse == 0) {
            if (op_type.compare("MUL") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
            else
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
        }
        else {
            if (op_type.compare("MUL") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
            else
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

        };

        return thrust::raw_pointer_cast(temp);

    }




    int_type* op(int_type* column1, int_type* column2, string op_type, int reverse)
    {

        thrust::device_ptr<int_type> temp = thrust::device_malloc<int_type>(mRecCount);
        thrust::device_ptr<int_type> dev_ptr1(column1);
        thrust::device_ptr<int_type> dev_ptr2(column2);

        if(reverse == 0) {
            if (op_type.compare("MUL") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<int_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<int_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<int_type>());
            else
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<int_type>());
        }
        else  {
            if (op_type.compare("MUL") == 0)
                thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
            else
                thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
        }

        return thrust::raw_pointer_cast(temp);

    }

    float_type* op(float_type* column1, float_type* column2, string op_type, int reverse)
    {

        thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
        thrust::device_ptr<float_type> dev_ptr1(column1);
        thrust::device_ptr<float_type> dev_ptr2(column2);

        if(reverse == 0) {
            if (op_type.compare("MUL") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<float_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<float_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<float_type>());
            else
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<float_type>());
        }
        else {
            if (op_type.compare("MUL") == 0)
                thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
            else
                thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());
        };
        return thrust::raw_pointer_cast(temp);
    }

    int_type* op(int_type* column1, int_type d, string op_type, int reverse)
    {
        thrust::device_ptr<int_type> temp = thrust::device_malloc<int_type>(mRecCount);
        thrust::fill(temp, temp+mRecCount, d);

        thrust::device_ptr<int_type> dev_ptr1(column1);

        if(reverse == 0) {
            if (op_type.compare("MUL") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<int_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<int_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<int_type>());
            else
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<int_type>());
        }
        else {
            if (op_type.compare("MUL") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
            else
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
        };
        return thrust::raw_pointer_cast(temp);
    }

    float_type* op(int_type* column1, float_type d, string op_type, int reverse)
    {
        thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
        thrust::fill(temp, temp+mRecCount, d);

        thrust::device_ptr<int_type> dev_ptr(column1);
        thrust::device_ptr<float_type> dev_ptr1 = thrust::device_malloc<float_type>(mRecCount);
        thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr1, long_to_float_type());

        if(reverse == 0) {
            if (op_type.compare("MUL") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
            else
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
        }
        else  {
            if (op_type.compare("MUL") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
            else
                thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

        };
        thrust::device_free(dev_ptr1);
        return thrust::raw_pointer_cast(temp);
    }


    float_type* op(float_type* column1, float_type d, string op_type,int reverse)
    {
        thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
        thrust::device_ptr<float_type> dev_ptr1(column1);

        if(reverse == 0) {
            if (op_type.compare("MUL") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::multiplies<float_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::plus<float_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::minus<float_type>());
            else
                thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::divides<float_type>());
        }
        else	{
            if (op_type.compare("MUL") == 0)
                thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
            else if (op_type.compare("ADD") == 0)
                thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
            else if (op_type.compare("MINUS") == 0)
                thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
            else
                thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

        };

        return thrust::raw_pointer_cast(temp);

    }


protected: // methods


    void initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, int_type Recs, char* file_name) // compressed data for DIM tables
    {
        mColumnCount = nameRef.size();
        type = new unsigned int[mColumnCount];
        cols = new unsigned int[mColumnCount];
        decimal = new bool[mColumnCount];
        unsigned int cnt, grp_count;
        file_p = NULL;
        FILE* f;
        char f1[100];

        fact_table = 0;
        mRecCount = Recs;
        load_file_name = file_name;
        //std::clock_t start1 = std::clock();


        for(unsigned int i=0; i < mColumnCount; i++) {

            columnNames[nameRef.front()] = i;
            cols[i] = colsRef.front();
            seq = 0;

            strcpy(f1, file_name);
            strcat(f1,".");
            char col_pos[3];
            itoaa(colsRef.front(),col_pos);
            strcat(f1,col_pos); // read the size of a segment


            f = fopen (f1 , "rb" );
            fread((char *)&cnt, 4, 1, f);
            //          cout << "creating host " << cnt << endl;

            if ((typeRef.front()).compare("int") == 0) {
                type[i] = 0;
                decimal[i] = 0;
                h_columns_int.push_back(thrust::host_vector<int_type>(cnt + 9));
                d_columns_int.push_back(thrust::device_vector<int_type>());
                type_index[i] = h_columns_int.size()-1;
            }
            else if ((typeRef.front()).compare("float") == 0) {
                type[i] = 1;
                decimal[i] = 0;
                h_columns_float.push_back(thrust::host_vector<float_type>(cnt + 9));
                d_columns_float.push_back(thrust::device_vector<float_type>());
                type_index[i] = h_columns_float.size()-1;
            }
            else if ((typeRef.front()).compare("decimal") == 0) {
                type[i] = 1;
                decimal[i] = 1;
                h_columns_float.push_back(thrust::host_vector<float_type>(cnt + 9));
                d_columns_float.push_back(thrust::device_vector<float_type>());
                type_index[i] = h_columns_float.size()-1;
            }
            else {
                type[i] = 2;
                decimal[i] = 0;
                fseeko(f, cnt*8 + 12, SEEK_SET);
                fread((char *)&grp_count, 4, 1, f);
                h_columns_cuda_char.push_back(new CudaChar(sizeRef.front(), Recs, 0, cnt*8 + 14*4 + grp_count*sizeRef.front()));
                type_index[i] = h_columns_cuda_char.size()-1;
            };

            fclose(f);
            nameRef.pop();
            typeRef.pop();
            sizeRef.pop();
            colsRef.pop();
        };
        //std::cout<< "create time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) <<'\n';
    };



    void initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, int_type Recs)
    {
        mColumnCount = nameRef.size();
        type = new unsigned int[mColumnCount];
        cols = new unsigned int[mColumnCount];
        decimal = new bool[mColumnCount];

        file_p = NULL;

        mRecCount = Recs;
        segCount = 1;

        for(unsigned int i=0; i < mColumnCount; i++) {

            columnNames[nameRef.front()] = i;
            cols[i] = colsRef.front();
            seq = 0;

            if ((typeRef.front()).compare("int") == 0) {
                type[i] = 0;
                decimal[i] = 0;
                h_columns_int.push_back(thrust::host_vector<int_type>());
                d_columns_int.push_back(thrust::device_vector<int_type>());
                type_index[i] = h_columns_int.size()-1;
            }
            else if ((typeRef.front()).compare("float") == 0) {
                type[i] = 1;
                decimal[i] = 0;
                h_columns_float.push_back(thrust::host_vector<float_type>());
                d_columns_float.push_back(thrust::device_vector<float_type>());
                type_index[i] = h_columns_float.size()-1;
            }
            else if ((typeRef.front()).compare("decimal") == 0) {
                type[i] = 1;
                decimal[i] = 1;
                h_columns_float.push_back(thrust::host_vector<float_type>());
                d_columns_float.push_back(thrust::device_vector<float_type>());
                type_index[i] = h_columns_float.size()-1;
            }

            else {
                type[i] = 2;
                decimal[i] = 0;
                h_columns_cuda_char.push_back(new CudaChar(sizeRef.front(), Recs, 1));
                type_index[i] = h_columns_cuda_char.size()-1;
            };
            nameRef.pop();
            typeRef.pop();
            sizeRef.pop();
            colsRef.pop();
        };
    };

    void initialize(unsigned int RecordCount, unsigned int ColumnCount)
    {
        mRecCount = RecordCount;
        mColumnCount = ColumnCount;

        type = new unsigned int[mColumnCount];
        cols = new unsigned int[mColumnCount];
        decimal = new bool[mColumnCount];
        seq = 0;

        for(unsigned int i =0; i < mColumnCount; i++)
            cols[i] = i;

    };


    void initialize(CudaSet* a, CudaSet* b, int_type Recs, queue<string> op_sel, queue<string> op_sel_as)
    {
        mRecCount = Recs;
        mColumnCount = op_sel.size();

        type = new unsigned int[mColumnCount];
        cols = new unsigned int[mColumnCount];
        decimal = new bool[mColumnCount];

        map<string,int>::iterator it;
        map<int,string> columnNames1;
        seq = 0;
        unsigned int i = 0;

        while(!op_sel_as.empty()) {
            columnNames[op_sel_as.front()] = i;
            op_sel_as.pop();
            i++;
        };

        if (Recs != 0) {
            unsigned int index;
            for(unsigned int i=0; i < mColumnCount; i++) {

                if((it = a->columnNames.find(op_sel.front())) !=  a->columnNames.end()) {
                    index = it->second;
                    cols[i] = i;
                    decimal[i] = a->decimal[i];

                    if ((a->type)[index] == 0)  {
                        d_columns_int.push_back(thrust::device_vector<int_type>());
                        h_columns_int.push_back(thrust::host_vector<int_type>());
                        type[i] = 0;
                        type_index[i] = h_columns_int.size()-1;
                    }
                    else if ((a->type)[index] == 1) {
                        d_columns_float.push_back(thrust::device_vector<float_type>());
                        h_columns_float.push_back(thrust::host_vector<float_type>());
                        type[i] = 1;
                        type_index[i] = h_columns_float.size()-1;
                    }
                    else {
                        h_columns_cuda_char.push_back(new CudaChar((a->h_columns_cuda_char[a->type_index[index]])->mColumnCount, Recs, 1));
                        type[i] = 2;
                        type_index[i] = h_columns_cuda_char.size()-1;
                    };
                }
                else {
                    it = b->columnNames.find(op_sel.front());
                    index = it->second;

                    cols[i] = i;
                    decimal[i] = b->decimal[index];

                    if ((b->type)[index] == 0) {
                        d_columns_int.push_back(thrust::device_vector<int_type>());
                        h_columns_int.push_back(thrust::host_vector<int_type>());
                        type[i] = 0;
                        type_index[i] = h_columns_int.size()-1;
                    }
                    else if ((b->type)[index] == 1) {
                        d_columns_float.push_back(thrust::device_vector<float_type>());
                        h_columns_float.push_back(thrust::host_vector<float_type>());
                        type[i] = 1;
                        type_index[i] = h_columns_float.size()-1;
                    }
                    else {
                        h_columns_cuda_char.push_back(new CudaChar((b->h_columns_cuda_char[a->type_index[index]])->mColumnCount, Recs, 1));
                        type[i] = 2;
                        type_index[i] = h_columns_cuda_char.size()-1;
                    };
                }
                op_sel.pop();
            };
        };
    }
};


int_type reverse_op(int_type op_type)
{
    if (op_type == 2) // >
        return 5;
    else if (op_type == 1)  // <
        return 6;
    else if (op_type == 6) // >=
        return 1;
    else if (op_type == 5)  // <=
        return 2;
    else return op_type;
}


size_t getFreeMem()
{
	size_t available, total;
    hipMemGetInfo(&available, &total); 
    return available;
} ;

void* LoadBuffers1(void* file_name)
{
    void* p = 0;
    LoadBuffers(file_name);
    return p;
}

void LoadBuffers(void* file_name)
{
    char str[100];
    char col_pos[3];
    unsigned int cnt;
    long long int lower_val, upper_val;
    map<unsigned int,unsigned int> counts;
    bool check_res = 0;
    FILE* f;


    while(runningRecs < totalRecs && !check_res) {
        for(unsigned int i = 0; i< th->mColumnCount; i++) {
            strcpy(str, (char*)file_name);
            strcat(str,".");
            itoaa(th->cols[i],col_pos);
            strcat(str,col_pos);

            if (th->filePointers.find(str) == th->filePointers.end())
                th->filePointers[str] = fopen(str, "rb");
            f = th->filePointers[str];

            if (th->type[i] == 0 || (th->type[i] == 1 && th->decimal[i]))  {
                fread(&cnt, 4, 1, f);
                counts[i] = cnt;
                fread(&lower_val, 8, 1, f);
                fread(&upper_val, 8, 1, f);
                unsigned int max_resize;
                if(cnt == 1)
                    max_resize = 2;
                else
                    max_resize = cnt;


                //cout << "segment upper lower " << upper_val << " " << lower_val << endl;
                if (th->type[i] == 0) {
                    if(cnt > th->h_columns_int[th->type_index[i]].size())
                        th->h_columns_int[th->type_index[i]].resize(max_resize);
                    (th->h_columns_int[th->type_index[i]])[0] = lower_val;
                    (th->h_columns_int[th->type_index[i]])[1] = upper_val;
                }
                else {

                    if(cnt > th->h_columns_float[th->type_index[i]].size())
                        th->h_columns_float[th->type_index[i]].resize(max_resize);
                    (th->h_columns_float[th->type_index[i]])[0] = ((float_type)lower_val)/100.0;
                    (th->h_columns_float[th->type_index[i]])[1] = ((float_type)upper_val)/100.0;

                };
            }
        };

        if(!top_type[th->name].empty()) {
            check_res = zone_map_check(top_type[th->name],top_value[th->name],top_nums[th->name],top_nums_f[th->name],th);

            //cout << "check result "	<< check_res << endl;
            if (!check_res) {   // do not process segment, move the pointers to the next segment
                runningRecs = runningRecs + th->maxRecs;
                if (runningRecs >= totalRecs) {
                    buffersEmpty = 1;
                    buffersLoaded = 1;
                    return;
                }
                else   {
                    // adjust file pointers

                    for(int z = 0; z< th->mColumnCount; z++) {

                        strcpy(str, (char*)file_name);
                        strcat(str,".");
                        itoaa(th->cols[z],col_pos);
                        strcat(str,col_pos);
                        f = th->filePointers[str];

                        if (th->type[z] == 0 || (th->type[z] == 1 && th->decimal[z]))
                            fseeko(f, counts[z]*8 + 44, SEEK_CUR);
                        else if (th->type[z] == 1 && !th->decimal[z])
                            fseeko(f, counts[z]*8 + 8, SEEK_CUR);
                        else {
                            unsigned int grp_count;
                            CudaChar *c = th->h_columns_cuda_char[th->type_index[z]];
                            fread(&cnt, 4, 1, f);
                            fseeko(f,cnt*8 + 8,SEEK_CUR);
                            fread(&grp_count, 4, 1, f);
                            fseeko(f,grp_count*c->mColumnCount,SEEK_CUR);
                        };
                    };
                };
            };
        }
        else
            check_res = 1;
    };


    for(unsigned int i = 0; i< th->mColumnCount; i++) {
        strcpy(str, (char*)file_name);
        strcat(str,".");
        itoaa(th->cols[i],col_pos);
        strcat(str,col_pos);

        f = th->filePointers[str];

        if (th->type[i] == 0)  {
            //fread(&cnt, 4, 1, f);
            //fread(&lower_val, 8, 1, f);
            //fread(&upper_val, 8, 1, f);
            fread(th->h_columns_int[th->type_index[i]].data(),counts[i]*8,1,f);
        }
        else if (th->type[i] == 1 && th->decimal[i]) {
            fread(th->h_columns_float[th->type_index[i]].data(),counts[i]*8,1,f);
        }
        else if (th->type[i] == 1 && !th->decimal[i]) {
            unsigned int grp_count;
            fread(&cnt, 4, 1, f);
            fread(th->h_columns_float[th->type_index[i]].data(),cnt*8,1,f);
            fread(&grp_count, 4, 1, f);
        }
        else {
            unsigned int grp_count;
            CudaChar *c = th->h_columns_cuda_char[th->type_index[i]];

            fread(&cnt, 4, 1, f);
            if(!c->compressed.size())
                c->compressed.resize(cnt*8);
            fread(c->compressed.data(),cnt*8,1,f);
            fread(&grp_count, 4, 1, f);
            fread(&grp_count, 4, 1, f);
            fread(&grp_count, 4, 1, f);
            for(unsigned int j = 0; j < c->mColumnCount; j++) {
                if(c->h_columns[j].size() < grp_count)
                    c->h_columns[j].resize(grp_count);
                fread(c->h_columns[j].data(),grp_count,1,f);
            };
        };
    };
    buffersLoaded = 1;
}


unsigned int findSegmentCount(char* file_name)
{
    unsigned int orig_recCount;
    unsigned int comp_type, cnt;

    FILE* f = fopen ( file_name , "rb" );
    if (f==NULL) {
        cout << "Cannot open file " << file_name << endl;
        exit (1);
    }
    fread(&cnt, 4, 1, f);
    fseeko(f, cnt*8 + 16, SEEK_CUR);
    fread(&comp_type, 4, 1, f);
    if(comp_type == 2)
        fread(&orig_recCount, 4, 1, f);
    else if(comp_type == 3)
        orig_recCount = cnt;
    else {
        fread(&orig_recCount, 4, 1, f);
        fread(&orig_recCount, 4, 1, f);
    };

    fclose(f);

    return orig_recCount;
};




void allocColumns(CudaSet* a, queue<string> fields)
{
    while(!fields.empty()) {
        if(setMap.count(fields.front()) > 0) {
            CudaSet *t = varNames[setMap[fields.front()]];
            unsigned int idx = t->columnNames[fields.front()];
            bool onDevice = 0;

            if(t->type[idx] == 0) {
                if(t->d_columns_int[t->type_index[idx]].size() > 0)
                    onDevice = 1;
            }
            else if(t->type[idx] == 1) {
                if(t->d_columns_float[t->type_index[idx]].size() > 0)
                    onDevice = 1;
            }
            else {
                if((t->h_columns_cuda_char[t->type_index[idx]])->d_columns[0].size() > 0)
                    onDevice = 1;
            };

            if (!onDevice)
                t->allocColumnOnDevice(t->columnNames[fields.front()], t->maxRecs);
        };
        fields.pop();
    };

}

unsigned int largest_prm(CudaSet* a, string field)
{
    unsigned int maxx = 0;

    for(unsigned int i = 0; i < a->prm_count[setMap[field]].size(); i++)
        if(maxx < a->prm_count[setMap[field]][i])
            maxx = a->prm_count[setMap[field]][i];
    return maxx;
};


void gatherColumns(CudaSet* a, CudaSet* t, string field, unsigned int segment)
{

    unsigned int tindex = t->columnNames[field];
    unsigned int idx = a->columnNames[field];

    //find the largest possible size of a gathered segment
    if(segment == 0) {
        unsigned int max_count = 0;

        if(a->prm.size() > 0) {
            for(unsigned int i = 0; i < a->segCount; i++)
                if(!a->isJoined) {  // alloc just 1 segment
                    if (a->prm_count[setMap[field]][i] > max_count)
                        max_count = a->prm_count[setMap[field]][i];
                }
                else {
                    // alloc entire result
                    max_count = max_count +  a->prm_count[setMap[field]][i];
                };
        }
        else {
            max_count = t->maxRecs;
        };
        a->allocColumnOnDevice(idx, max_count);
    };


    if(!a->isJoined) {
        if(a->prm.size() > 0) {

            unsigned int g_size = a->prm_count[setMap[field]][segment];

            //cout << "largest prm " << largest_prm(a, field) << endl;
            if(a->prm_d.size() == 0) // find the largest prm segment
                a->prm_d.resize(largest_prm(a, field));

            if(curr_segment != segment) {
                std::clock_t start2 = std::clock();

                hipMemcpy((void**)(thrust::raw_pointer_cast(a->prm_d.data())), (void**)a->prm[setMap[field]][segment],
                           4*g_size, hipMemcpyHostToDevice);
                curr_segment = segment;
            };

            if(t->type[tindex] == 0)
                thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                               t->d_columns_int[t->type_index[tindex]].begin(), a->d_columns_int[a->type_index[idx]].begin());
            else if(t->type[tindex] == 1)
                thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                               t->d_columns_float[t->type_index[tindex]].begin(), a->d_columns_float[a->type_index[idx]].begin());
            else
                for(unsigned int j=0; j < (t->h_columns_cuda_char[t->type_index[tindex]])->mColumnCount; j++)
                    thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                                   (t->h_columns_cuda_char[t->type_index[tindex]])->d_columns[j].begin(),
                                   (a->h_columns_cuda_char[a->type_index[idx]])->d_columns[j].begin());
            a->mRecCount = g_size;
        }
        else {

            if(t->type[tindex] == 0) {
                thrust::copy(t->d_columns_int[t->type_index[tindex]].begin(), t->d_columns_int[t->type_index[tindex]].end(),
                             a->d_columns_int[a->type_index[idx]].begin());
                a->mRecCount = 	t->d_columns_int[t->type_index[tindex]].end() - t->d_columns_int[t->type_index[tindex]].begin();
            }
            else if(t->type[tindex] == 1) {
                thrust::copy(t->d_columns_float[t->type_index[tindex]].begin(), t->d_columns_float[t->type_index[tindex]].end(),
                             a->d_columns_float[a->type_index[idx]].begin());
                a->mRecCount = t->d_columns_float[t->type_index[tindex]].end() - t->d_columns_float[t->type_index[tindex]].begin();
            }
            else {
                for(unsigned int j=0; j < (t->h_columns_cuda_char[t->type_index[tindex]])->mColumnCount; j++) {
                    thrust::copy((t->h_columns_cuda_char[t->type_index[tindex]])->d_columns[j].begin(),(t->h_columns_cuda_char[t->type_index[tindex]])->d_columns[j].end(),
                                 (a->h_columns_cuda_char[a->type_index[idx]])->d_columns[j].begin());
                    a->mRecCount = (t->h_columns_cuda_char[t->type_index[tindex]])->d_columns[j].end() - (t->h_columns_cuda_char[t->type_index[tindex]])->d_columns[j].begin();
                };
            };

        };
    }
    else {
        // modify prm

        if(a->prm_d.size() == 0) // find the largest prm segment
            a->prm_d.resize(largest_prm(a, field));
        unsigned int g_size = a->prm_count[setMap[field]][0];
        hipMemcpy((void**)(thrust::raw_pointer_cast(a->prm_d.data())), (void**)a->prm[setMap[field]][0],
                   4*g_size, hipMemcpyHostToDevice);

        if (segment != 0)
            thrust::transform(a->prm_d.begin(), a->prm_d.begin() + g_size,
                              a->prm_d.begin(), _1 - t->maxRecs);

        if(t->type[tindex] == 0)
            thrust::gather_if(a->prm_d.begin(), a->prm_d.begin()+g_size, a->prm_d.begin(),
                              t->d_columns_int[t->type_index[tindex]].begin(), a->d_columns_int[a->type_index[idx]].begin(), _1 < t->maxRecs );
        else if(t->type[tindex] == 1)
            thrust::gather_if(a->prm_d.begin(), a->prm_d.begin()+g_size, a->prm_d.begin(),
                              t->d_columns_float[t->type_index[tindex]].begin(), a->d_columns_float[a->type_index[idx]].begin(), _1 < t->maxRecs);
        else
            for(unsigned int j=0; j < (t->h_columns_cuda_char[t->type_index[tindex]])->mColumnCount; j++)
                thrust::gather_if(a->prm_d.begin(), a->prm_d.begin()+g_size, a->prm_d.begin(),
                                  (t->h_columns_cuda_char[t->type_index[tindex]])->d_columns[j].begin(),
                                  (a->h_columns_cuda_char[a->type_index[idx]])->d_columns[j].begin(), _1 < t->maxRecs);

    };
}

void gatherColumnsJoin(CudaSet* a, CudaSet* t, string field, unsigned int segment, thrust::device_ptr<unsigned int>& m, unsigned int count )
{

    unsigned int tindex = t->columnNames[field];
    unsigned int idx = a->columnNames[field];

    //cout << "gathering " << field << " " << setMap[field] << " " << tindex << " " << idx << " " << a->segCount << endl;


    if(!a->isJoined) {
        if(a->prm_d.size() == 0) // find the largest prm segment
            a->prm_d.resize(largest_prm(a, field));
        unsigned int g_size = a->prm_count[setMap[field]][segment];
        hipMemcpy((void**)(thrust::raw_pointer_cast(a->prm_d.data())), (void**)a->prm[setMap[field]][segment],
                   4*g_size, hipMemcpyHostToDevice);

        thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                       t->d_columns_int[t->type_index[tindex]].begin(), m + count);
    }
    else {
        // modify prm
        if(a->prm_d.size() == 0) // find the largest prm segment
            a->prm_d.resize(largest_prm(a, field));
        unsigned int g_size = a->prm_count[setMap[field]][0];
        if(segment == 0)
            hipMemcpy((void**)(thrust::raw_pointer_cast(a->prm_d.data())), (void**)a->prm[setMap[field]][0],
                       4*g_size, hipMemcpyHostToDevice);

        if (segment != 0)
            thrust::transform(a->prm_d.begin(), a->prm_d.begin() + g_size,
                              a->prm_d.begin(), _1 - t->maxRecs);

        thrust::gather_if(a->prm_d.begin(), a->prm_d.begin() + g_size, a->prm_d.begin(),
                          t->d_columns_int[t->type_index[tindex]].begin(), m, _1 < t->maxRecs );

    };

    //a->mRecCount = a->prm[setMap[field]][segment].end() - a->prm[setMap[field]][segment].begin();
}


void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment)
{
    set<string> uniques;
    CudaSet *t;


    while(!fields.empty()) {
        if (uniques.count(fields.front()) == 0 && setMap.count(fields.front()) > 0)	{
            t = varNames[setMap[fields.front()]];

            if (!a->isJoined) {
                t->CopyColumnToGpu(t->columnNames[fields.front()], segment); // segment i
                if (a != t)  {
                    gatherColumns(a, t, fields.front(), segment);
                };
            }
            else {
                //for all segments do copy and gather
                unsigned int tindex = t->columnNames[fields.front()];
                unsigned int idx = a->columnNames[fields.front()];
                string field = fields.front();

                if(a->prm_d.size() == 0) // find the largest prm segment
                    a->prm_d.resize(largest_prm(a, field));
                unsigned int g_size = a->prm_count[setMap[field]][0];

                hipMemcpy((void**)(thrust::raw_pointer_cast(a->prm_d.data())), (void**)a->prm[setMap[field]][0], 4*g_size, hipMemcpyHostToDevice);


                for(unsigned int i = 0; i < t->segCount; i++) {
                    t->CopyColumnToGpu(t->columnNames[field], i); // segment i

                    if (i != 0) {
                        thrust::transform(a->prm_d.begin(), a->prm_d.begin() + g_size,
                                          a->prm_d.begin(), _1 - t->maxRecs);
                    }
                    else  {
                        a->allocColumnOnDevice(a->columnNames[field], g_size);
                    };


                    if(t->type[tindex] == 0)
                        thrust::gather_if(a->prm_d.begin(), a->prm_d.begin() + g_size, a->prm_d.begin(),
                                          t->d_columns_int[t->type_index[tindex]].begin(), a->d_columns_int[a->type_index[idx]].begin(), _1 < t->maxRecs );
                    else if(t->type[tindex] == 1)
                        thrust::gather_if(a->prm_d.begin(), a->prm_d.begin() + g_size, a->prm_d.begin(),
                                          t->d_columns_float[t->type_index[tindex]].begin(), a->d_columns_float[a->type_index[idx]].begin(), _1 < t->maxRecs);
                    else
                        for(unsigned int j=0; j < (t->h_columns_cuda_char[t->type_index[tindex]])->mColumnCount; j++)
                            thrust::gather_if(a->prm_d.begin(), a->prm_d.begin() + g_size, a->prm_d.begin(),
                                              (t->h_columns_cuda_char[t->type_index[tindex]])->d_columns[j].begin(),
                                              (a->h_columns_cuda_char[a->type_index[idx]])->d_columns[j].begin(), _1 < t->maxRecs);





                };

                if (t->segCount != 1)
                    thrust::transform(a->prm_d.begin(), a->prm_d.begin() + g_size,
                                      a->prm_d.begin(), _1 + (t->maxRecs*(t->segCount-1)));

            };
            uniques.insert(fields.front());
        };
        fields.pop();
    };
}


void copyGatherJoin(CudaSet* a, thrust::device_ptr<unsigned int>& m, string field, unsigned int segment, unsigned int& cnt )
{
    CudaSet *t;

    t = varNames[setMap[field]];
    unsigned int tindex = t->columnNames[field];

    if (!a->isJoined) {
        t->CopyColumnToGpu(t->columnNames[field], segment); // segment i
        if(a != t) {
            //gatherColumns(a, t, field, segment);
            gatherColumnsJoin(a, t, field, segment, m, cnt);
            cnt = cnt + a->prm_count[setMap[field]][segment];
        }
        else
            cnt = t->mRecCount;
    }
    else {
        //for all segments do copy and gather
        for(unsigned int i = 0; i < t->segCount; i++) {
            t->CopyColumnToGpu(t->columnNames[field], i); // segment i
            gatherColumnsJoin(a, t, field, i, m, cnt);
        };
        // transform prm back
        //a->prm_d = a->prm[setMap[field]][0];
        if(a->prm_d.size() == 0) // find the largest prm segment
            a->prm_d.resize(largest_prm(a, field));
        unsigned int g_size = a->prm_count[setMap[field]][0];
        hipMemcpy((void**)(thrust::raw_pointer_cast(a->prm_d.data())), (void**)a->prm[setMap[field]][0], 4*g_size, hipMemcpyHostToDevice);

        if (t->segCount != 1)
            thrust::transform(a->prm_d.begin(), a->prm_d.begin() + g_size,
                              a->prm_d.begin(), _1 + (t->maxRecs*(t->segCount-1)));
        cnt = cnt + g_size;
    };
}

